#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <hip/hip_runtime.h>

// CUDA-ядро для умножения матриц на GPU
__global__ void multiplyMatricesGPUKernel(const int* A, const int* B, int* C, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < n && col < n) {
        int sum = 0;
        for (int k = 0; k < n; ++k) {
            sum += A[row * n + k] * B[k * n + col];
        }
        C[row * n + col] = sum;
    }
}

// Функция для выполнения умножения матриц на GPU
void multiplyMatricesGPU(const int* A, const int* B, int* C, int n) {
    // Выделение памяти на GPU
    int* d_A, * d_B, * d_C;
    hipMalloc(&d_A, n * n * sizeof(int));
    hipMalloc(&d_B, n * n * sizeof(int));
    hipMalloc(&d_C, n * n * sizeof(int));

    // Копирование данных с CPU на GPU
    hipMemcpy(d_A, A, n * n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, n * n * sizeof(int), hipMemcpyHostToDevice);

    // Определение размеров блоков и сетки
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((n + threadsPerBlock.x - 1) / threadsPerBlock.x, (n + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // Запуск CUDA-ядра
    multiplyMatricesGPUKernel << <blocksPerGrid, threadsPerBlock >> > (d_A, d_B, d_C, n);
    hipDeviceSynchronize();

    // Копирование результата обратно на CPU
    hipMemcpy(C, d_C, n * n * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}
